#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""

extern "C"{
#include "ppmFile.h"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Keep x and y values within the boundaries of the image
__device__ __host__
int checkBoundsHeight(int val,int height) {

  if (val < 0){
    val = 0;
  } else if (val > height-1){
    val = height-1;
  }

  return val;
}

// Keep x and y values within the boundaries of the image
__device__ __host__
int checkBoundsWidth(int val, int width){

  if (val < 0){
    val = 0;
  } else if (val > width-1){
    val = width-1;
  }

  return val;
}

__device__ __host__
void ImageSetPixelDevice(Image *image, int x, int y, int chan, unsigned char val){
  int offset = (y * image->width + x) * 3 + chan;

  image->data[offset] = val;
}

__device__ __host__
unsigned  char ImageGetPixelDevice(Image *image, int x, int y, int chan){
  int offset = (y * image->width + x) * 3 + chan;

  return image->data[offset];
}

// Calculates new image color for blurring
__device__ __host__
int calcBlurColour(Image *imageFile, int minX, int maxX, int minY, int maxY, int channel){
  //pixelsAdded = (maxX - minX)*(maxY-minY);
  int pixelsAdded = 0;
  long val = 0;

  for (int x = minX; x <= maxX; x++){
    for (int y = minY; y <= maxY; y++){
      val = val + ImageGetPixelDevice(imageFile,x,y,channel);
      pixelsAdded++;
    }
  }

  if (pixelsAdded <= 0){
    pixelsAdded = 1;
  }

  val = val/pixelsAdded;

  return val;
}

__global__
void blurImage(unsigned char *data, unsigned char *blurData, int height, int width, int r,int totalThreads){

  int myID = (blockIdx.z * gridDim.x * gridDim.y +
              blockIdx.y * gridDim.x +
              blockIdx.x) * blockDim.x +
              threadIdx.x;
  int minX;
  int maxX;
  int minY;
  int maxY;

  unsigned char newR = 0;

  unsigned char newG = 0;
  unsigned char newB = 0;

  Image *image = (Image *) malloc(sizeof(Image));
  image->data   = data;
  image->width  = width;
  image->height = height;

  for (int y = 0; y < height; y++){
    for (int x = myID; x < width; x = x + totalThreads){
      image->data = data;
      // Calculates new bounds for the blurring of the pixel based on the blur radius
      minX = checkBoundsWidth(x - r,image->width);
      maxX = checkBoundsWidth(x + r,image->width);
      minY = checkBoundsHeight(y - r,image->height);
      maxY = checkBoundsHeight(y + r,image->height);

      newR = calcBlurColour(image, minX, maxX, minY, maxY,0);
      newG = calcBlurColour(image, minX, maxX, minY, maxY,1);
      newB = calcBlurColour(image, minX, maxX, minY, maxY,2);

      image->data = blurData;

      ImageSetPixelDevice(image, x, y, 0, newR);
      ImageSetPixelDevice(image, x, y, 1, newG);
      ImageSetPixelDevice(image, x, y, 2, newB);
    }
  }
}

__global__
void pixelateImage(unsigned char *data, unsigned char *blurData, int height, int width, int r,int totalThreads){

  int myID = (blockIdx.z * gridDim.x * gridDim.y +
              blockIdx.y * gridDim.x +
              blockIdx.x) * blockDim.x +
              threadIdx.x;
  int minX;
  int maxX;
  int minY;
  int maxY;

  int spacing = r*2;

  unsigned char newR = 0;

  unsigned char newG = 0;
  unsigned char newB = 0;

  Image *image = (Image *) malloc(sizeof(Image));
  image->data   = data;
  image->width  = width;
  image->height = height;

  for (int y = r; y < height; y = y + spacing){
    for (int x = r; x < width; x = x + spacing){
      image->data = data;
      // Calculates new bounds for the blurring of the pixel based on the blur radius
      minX = checkBoundsWidth(x - r,image->width);
      maxX = checkBoundsWidth(x + r,image->width);
      minY = checkBoundsHeight(y - r,image->height);
      maxY = checkBoundsHeight(y + r,image->height);

      newR = calcBlurColour(image, minX, maxX, minY, maxY,0);
      newG = calcBlurColour(image, minX, maxX, minY, maxY,1);
      newB = calcBlurColour(image, minX, maxX, minY, maxY,2);

      image->data = blurData;

      for (int x = minX; x < maxX; x++){
        for (int y = minY; y < maxY; y++){
          // Blurs 3 channels (r,g,b) of pixel
          ImageSetPixelDevice(image,x,y,0,newR);
          ImageSetPixelDevice(image,x,y,1,newG);
          ImageSetPixelDevice(image,x,y,2,newB);
        }
      }
    }
  }
}

int main(int argc, char**argv){
  Image *imageFile, *outputImage;
  unsigned char *data,*blurData;
  int r;

  int blockHeight = 1;
  int blockWidth = 1024;

  int gridHeight = 1;
  int gridWidth = 1;
  int gridLength = 1;

  int totalThreads = blockHeight*blockWidth*gridHeight*gridWidth*gridLength;

  dim3 block(blockHeight,blockWidth);
  dim3 grid(gridHeight,gridWidth,gridLength);

  // For clocking time
  time_t start,end1,end2;


  imageFile = ImageRead(argv[2]);
  outputImage = ImageRead(argv[2]);
  r = strtol(argv[1],NULL,10);
  const size_t imageSize = sizeof(int)*2 + imageFile->height*imageFile->width*3;

  data = (unsigned char*) malloc(imageFile->height*imageFile->width*3);
  blurData = (unsigned char*) malloc(imageFile->height*imageFile->width*3);

  // Allocate unified memory for cpu and gpu
  gpuErrchk(hipMallocManaged(&data,imageFile->height*imageFile->width*3));
  gpuErrchk(hipMallocManaged(&blurData,imageFile->height*imageFile->width*3));
  memcpy(data,imageFile->data,imageFile->height*imageFile->width*3);
  //memcpy(blurData,imageFile->data,imageFile->height*imageFile->width*3);

  printf("Blurring Image based on GPU...\n");

  // Start Timer
  start = clock();
  printf("%s\n",argv[4]);
  if (strcmp(argv[4],"blur") == 0){
    blurImage<<<gridHeight,blockHeight*blockWidth>>>(data,blurData,imageFile->height,imageFile->width,r,totalThreads);
    hipDeviceSynchronize();
  } else {
    pixelateImage<<<gridHeight,blockHeight*blockWidth>>>(data,blurData,imageFile->height,imageFile->width,r,totalThreads);
    hipDeviceSynchronize();
  }

  // References blurred data to output image
  outputImage->data = blurData;

  // End Timer
  end1 = clock() - start;

  ImageWrite(outputImage,argv[3]);
  printf("Image: %s blurred, saved as: %s\n",argv[2],argv[3]);
  printf("Clock ticks: %li\n",end1);

  //endTime = MPI_Wtime();
  //totalTime = endTime - startTime;
  //printf("Took %f seconds\n", totalTime);

  hipFree(data);
  hipFree(blurData);
  return 0;
}
